#include "hip/hip_runtime.h"
/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "cuda_utils.h"
/* #include "../node/n_step.h" */

namespace FT{
   		
    __global__ void Step(float * x, size_t idx, size_t N)
    {                    
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
        {
            if (x[(idx-1)*N+i] > 0 )
                x[(idx-1)*N+i] = 1.0 ; 
            else
                x[(idx-1)*N+i] = 0.0 ;
        }
        return;
    }
    void GPU_Step(float * x, size_t idx, size_t N)
    {
        GPU_Step<<< DIM_GRID, DIM_BLOCK, omp_get_thread_num() >>>(float * x, size_t idx, size_t N);
    }
    /// Evaluates the node and updates the stack states. 
    /* void NodeStep::evaluate(const MatrixXd& X, const VectorXd& y, vector<ArrayXd>& stack_f, */ 
    /*         vector<ArrayXb>& stack_b) */
    /* { */
    /*     ArrayXd x1 = stack_f.back(); stack_f.pop_back(); */
    /*     // evaluate on the GPU */
    /*     ArrayXd result = ArrayXd(x1.size()); */
    /*     size_t N = result.size(); */
    /*     double * dev_res; */
    /*     int numSMs; */
    /*     hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0); */
    /*     // allocate device arrays */
    /*     double * dev_x1 ; */ 
    /*     HANDLE_ERROR(hipMalloc((void **)& dev_x1, sizeof(double)*N)); */
    /*     HANDLE_ERROR(hipMalloc((void **)&dev_res, sizeof(double)*N)); */
    /*     // Copy to device */
    /*     HANDLE_ERROR(hipMemcpy(dev_x1, x1.data(), sizeof(double)*N, hipMemcpyHostToDevice)); */

    /*     Step<<< 32*numSMs, 128 >>>(dev_x1, dev_res, N); */
       
    /*     // Copy to host */
    /*     HANDLE_ERROR(hipMemcpy(result.data(), dev_res, sizeof(double)*N, hipMemcpyDeviceToHost)); */
        
    /*     stack_f.push_back(limited(result)); */
    /*     // Free memory */
    /*     hipFree(dev_x1); hipFree(dev_res); */
    /* } */

}	


