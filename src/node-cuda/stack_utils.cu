/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "stack_utils.h"
#include "error_handling.h"
// stack utils 
namespace FT{
    void dev_allocate(float *& f, bool *& b, size_t Sizef, size_t Sizeb)
    {
        HANDLE_ERROR(hipMalloc((void **)& f, sizeof(float)*Sizef));
        HANDLE_ERROR(hipMalloc((void **)& b, sizeof(bool)*Sizeb));
	HANDLE_ERROR(hipDeviceSynchronize());
        /* std::cout << "allocated " << sizeof(float)*Sizef << " bytes at loc " << f << " for stack.f\n"; */
        /* std::cout << "allocated " << sizeof(bool)*Sizeb << " bytes at loc " << b << " for stack.b\n"; */
    }

    void copy_from_device(float * dev_f, float * host_f, bool * dev_b, bool * host_b, size_t Sizef, 
                          size_t Sizeb)
    {
        /* std::cout << "dev_f: " << dev_f << "\nhost_f: " << host_f << "\nSizef: " << Sizef << "\n"; */
                  
        HANDLE_ERROR(hipMemcpy(host_f, dev_f, sizeof(float)*Sizef, hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(host_b, dev_b, sizeof(bool)*Sizeb,  hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipDeviceSynchronize());
    }

    void free_device(float * dev_f, bool * dev_b)
    {
        // Free memory
        hipFree(dev_f); 
        hipFree(dev_b);         
	HANDLE_ERROR(hipDeviceSynchronize());
    }
}
