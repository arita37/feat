#include "hip/hip_runtime.h"
/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "error_handling.h"
#include "cuda_utils.h"
/* #include "../node/n_ifthenelse.h" */

namespace FT{
   		
    __global__ void IfThenElse(bool * b, float * x, size_t idxb, size_t idxf, size_t N)
    {                    
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
        {
            if (b[(idxb-1)*N+i])
                x[(idxf-2)*N+i] = x[(idxf-1)*N+i];
        }
        return;
    }
    void GPU_IfThenElse(float * x, bool *b ,size_t idxb, size_t idxf, size_t N)
    {
        IfThenElse<<< DIM_GRID, DIM_BLOCK >>>(b, x, idxb, idxf, N);
    }
    /// Evaluates the node and updates the stack states. 
    /* void NodeIfThenElse::evaluate(const MatrixXd& X, const VectorXd& y, vector<ArrayXd>& stack_f, */ 
    /*         vector<ArrayXb>& stack_b) */
    /* { */
    /*     ArrayXd x2 = stack_f.back(); stack_f.pop_back(); */
    /*     ArrayXd x1 = stack_f.back(); stack_f.pop_back(); */
    /*     // evaluate on the GPU */
    /*     ArrayXd result = ArrayXd(x1.size()); */
    /*     size_t N = result.size(); */
    /*     double * dev_res; */
    /*     int numSMs; */
    /*     hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0); */
    /*     // allocate device arrays */
    /*     double * dev_x1, * dev_x2 ; */ 
    /*     bool * dev_b1; */

    /*     HANDLE_ERROR(hipMalloc((void **)& dev_b1, sizeof(bool)*N)); */
    /*     HANDLE_ERROR(hipMalloc((void **)& dev_x1, sizeof(double)*N)); */
    /*     HANDLE_ERROR(hipMalloc((void **)& dev_x2, sizeof(double)*N)); */
    /*     HANDLE_ERROR(hipMalloc((void **)&dev_res, sizeof(double)*N)); */
    /*     // Copy to device */
    /*     HANDLE_ERROR(hipMemcpy(dev_x1, x1.data(), sizeof(double)*N, hipMemcpyHostToDevice)); */
    /*     HANDLE_ERROR(hipMemcpy(dev_x2, x2.data(), sizeof(double)*N, hipMemcpyHostToDevice)); */

    /*     IfThenElse<<< 32*numSMs, 128 >>>(dev_b1, dev_x1, dev_x2, dev_res, N); */
       
    /*     // Copy to host */
    /*     HANDLE_ERROR(hipMemcpy(result.data(), dev_res, sizeof(double)*N, hipMemcpyDeviceToHost)); */
        
    /*     stack_f.push_back(result); */
    /*     // Free memory */
    /*     hipFree(dev_x1); hipFree(dev_x2); hipFree(dev_res); */
    /* } */

}	


