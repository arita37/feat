/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "cuda_utils.h"

static void Initialize()
{
    hipDeviceGetAttribute(&NUM_SMS, hipDeviceAttributeMultiprocessorCount, 0); 
    DIM_GRID = 32*NUM_SMS;
    DIM_BLOCK = 128; 

}
static void HandleError( hipError_t err, const char *file, int line )
{
	// CUDA error handeling from the "CUDA by example" book
	if (err != hipSuccess)
    {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}


