/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "cuda_utils.h"

int NUM_SMS = 32; 
int DIM_GRID = 1024; 
int DIM_BLOCK = 128; 


void FT::initialize_cuda()
{
    hipDeviceGetAttribute(&NUM_SMS, hipDeviceAttributeMultiprocessorCount, 0); 
    DIM_GRID = 32*NUM_SMS;
    DIM_BLOCK = 128; 
}

void FT::choose_gpu()
{
    //#pragma omp critical
	//{
		int n_gpus; 
    	hipGetDeviceCount(&n_gpus);
    	int device = omp_get_thread_num() % n_gpus ; 
    	hipSetDevice(device);
		//hipDeviceSynchronize();
		//hipSetDevice(1);
	//} 

}


