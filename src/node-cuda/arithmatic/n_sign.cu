#include "hip/hip_runtime.h"
/* FEAT
copyright 2017 William La Cava
license: GNU/GPL v3
*/
#include "../error_handling.h"
#include "../cuda_utils.h"

namespace FT{
   		
    __global__ void Sign( float * x, size_t idx, size_t N, float W0)
    {                    
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
        {
            float res = W0*x[(idx-1)*N+i];
            if (res > 0 )
                x[(idx-1)*N+i] = 1.0 ; 
            else if (res == 0)
                x[(idx-1)*N+i] = 0.0; 
            else
                x[(idx-1)*N+i] = -1.0 ;
        }
        return;
    }
    void GPU_Sign( float * x, size_t idx, size_t N, float W0)
    {
        Sign<<< DIM_GRID, DIM_BLOCK >>>(x, idx, N, W0);
    }
    /// Evaluates the node and updates the stack states. 
    /* void NodeSign::evaluate(const MatrixXd& X, const VectorXd& y, vector<ArrayXd>& stack_f, */ 
    /*         vector<ArrayXb>& stack_b) */
    /* { */
    /*     ArrayXd x1 = stack_f.back(); stack_f.pop_back(); */
    /*     // evaluate on the GPU */
    /*     ArrayXd result = ArrayXd(x1.size()); */
    /*     size_t N = result.size(); */
    /*     double * dev_res; */
    /*     int numSMs; */
    /*     hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0); */
    /*     // allocate device arrays */
    /*     double * dev_x1 ; */ 
    /*     HANDLE_ERROR(hipMalloc((void **)& dev_x1, sizeof(double)*N)); */
    /*     HANDLE_ERROR(hipMalloc((void **)&dev_res, sizeof(double)*N)); */
    /*     // Copy to device */
    /*     HANDLE_ERROR(hipMemcpy(dev_x1, x1.data(), sizeof(double)*N, hipMemcpyHostToDevice)); */

    /*     Sign<<< 32*numSMs, 128 >>>(dev_x1, dev_res, N); */
       
    /*     // Copy to host */
    /*     HANDLE_ERROR(hipMemcpy(result.data(), dev_res, sizeof(double)*N, hipMemcpyDeviceToHost)); */
        
    /*     stack_f.push_back(limited(result)); */
    /*     // Free memory */
    /*     hipFree(dev_x1); hipFree(dev_res); */
    /* } */

}	


